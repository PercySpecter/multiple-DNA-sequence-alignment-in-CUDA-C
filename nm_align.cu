

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 30

//typedef long long int ll;

__global__ void align(char *key , char *s , int *scores , int n , int num)
{
	int GP = -1 , MR = 1;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < num)
	{
		int i , j , k , tmp;
		int nm[N + 1][N + 1];
		
		char r1[2*N+2] , r2[2*N+2];
		for (i = 0; i <= n; i++)
		{
			nm[0][i] = GP * i;
			nm[i][0] = GP * i;
		}
		
		for (i = 1; i <= n; i++)
		{
			for (j = 1; j <= n; j++)
			{
				if(key[i-1] == s[n*index + j-1])
					nm[i][j] = nm[i-1][j-1] + MR;
				else	
				{
					if(nm[i-1][j] <= nm[i][j-1])
						nm[i][j] = nm[i][j-1] + GP;
					else
						nm[i][j] = nm[i-1][j] + GP;
				}
			}
		}
		
		/* for (i = 0; i <= n; i++)
		{
			for (j = 0; j <= n; j++)
			{
				printf("%d " , nm[i][j]);
			}
			printf("\n");
		} */
		
		i = n , j = n , k = 0;
		//for(int cnt = 1; cnt <= 30 && i > 0 && j > 0; cnt++)
		while(i > 0 && j > 0)
		{
			//if(index == 3)printf("**%d %d % d\n" , cnt , i , j);
			tmp = nm[i-1][j-1] > nm[i][j-1] ? (nm[i-1][j-1] > nm[i-1][j] ? nm[i-1][j-1] : nm[i-1][j]) : (nm[i][j-1] > nm[i-1][j] ? nm[i][j-1] : nm[i-1][j]);
			if(tmp == nm[i-1][j-1] || key[i] == s[n*index + j-1])
			{
				r1[k] = key[i-1];
				r2[k] = s[n*index + j-1];
				i--;
				j--;
			}
			else if(tmp == nm[i][j-1])
			{
				r1[k] = '-';
				r2[k] = s[n*index + j-1];
				j--;
			}
			else if(tmp == nm[i-1][j])
			{
				r1[k] = key[i-1];
				r2[k] = '-';
				i--;
			}
			k++;
		}
		for(i = 0; i < k/2; i++)
		{
			r1[i] = (r1[i] + r1[k-i-1]) - (r1[k-i-1] = r1[i]);
			r2[i] = (r2[i] + r2[k-i-1]) - (r2[k-i-1] = r2[i]);
		}
		r1[k] = '\0';
		r2[k] = '\0';
		printf("\nAlignment #%d :\n-------------------\nKey:\n%s\nQuery:\n%s\n" , index+1 , r1 , r2);
		int score = 0;
		for(i = 0; i < k; i++)
		{
			if(r1[k] == '-' || r2[k] == '-')
				score += GP;
			else if(r1[i] == r2[i])
				score += MR;
			else
				score += GP;
		}
		scores[index] = score;
	}
}

int main(int argc, char** argv)
{
	int size = sizeof(int);
	int THREADS = 1024;
	
	freopen(argv[1] , "r", stdin);
	freopen(argv[2] , "w", stdout);
	
	int *host_scores , *scores;
	int i , num , n;
	
	//printf("Enter size:");
	scanf("%d" , &n);
	//printf("Enter number of queries:");
	scanf("%d" , &num);
	
	int m = num < THREADS ? num : THREADS;
	
	char *host_key = (char *)malloc(n);
	char *tmp = (char *)malloc(n);
	char *host_q = (char *)malloc(num * n + 2);
	char *key , *q;
	
	//printf("Enter key:");
	scanf("%s" , host_key);
	//printf("Enter the queries:");
	for(i = 0; i <num; i++)
	{
		if(i == 0)
			scanf("%s" , host_q);
		else
		{
			scanf("%s" , tmp);
			strcat(host_q , tmp);
		}
	}
	
	host_scores = (int *)malloc(size * num);
	
	hipMalloc((void **)&scores , num * size);
	hipMalloc((void **)&key , n);
	hipMalloc((void **)&q , n * num + 2);
	hipMemcpy(key , host_key , n , hipMemcpyHostToDevice);
	hipMemcpy(q , host_q , n * num + 2 , hipMemcpyHostToDevice);
	
	align <<<(n + m - 1) / m , m>>> (key , q , scores , n , num);

	hipMemcpy(host_scores , scores , size * num , hipMemcpyDeviceToHost);

	printf("\n\nAlignment Scores:\n----------------------------\n");
	for(i = 0; i < num; i++)
		printf("Query #%d : %d\n" , i+1 , host_scores[i]);
	hipFree(key);
	hipFree(q);
	hipFree(scores);
	return 0;
}