/**
	@author Kinjal Ray
	Date 06.01.19
	DNA Sequence alignment using Pointing Matrix
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 300

//typedef long long int ll;

__global__ void align(char *key , char *s , int *scores , int n , int num)
{
	const int GP = -1 , MR = 1;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < num)
	{
		int c_row = 1 , o_row , c_score;
		int sm[2][N+1];
		int pm[N+1][N+1];
		char r1[2*N+2] , r2[2*N+2];
		int i , j;
		/*Initialising Scoring Matrix*/
		for (i = 0; i <= n; i++)
		{
			sm[0][i] = GP * i;
			pm[0][i] = 1;
			pm[i][0] = 3;
		}
		pm[0][0] = 0;
		
		/*Filling the Pointing Matrix*/
		for(i = 1; i <= n; i++)
		{
			o_row = 1 - c_row;
			for(j = 1; j <= n; j++)
			{
				if(key[i-1] == s[n*index + j-1])
					c_score = MR;
				else
					c_score = GP;
				int max = sm[o_row][j-1] + c_score;
				int ptr = 2;
				if(max < sm[o_row][j] + GP)
				{
					max = sm[o_row][j] + GP;
					ptr = 3;
				}
				if(max < sm[c_row][j-1] + GP)
				{
					max = sm[c_row][j-1] + GP;
					ptr = 1;
				}
				sm[c_row][j] = max;
				pm[i][j] = ptr;
			}
			c_row = o_row;
		}
		
		/*for(i = 0; i <= n; i++)
		{
			for(j = 0; j <= n; j++)
			{
				printf("%d " , pm[i][j]);
			}
			printf("\n");
		}*/
		
		/*Sequence Alignment using Pointing Matrix*/
		int score = 0 , row = n , col = n;
		j = 0;
		//printf("PM : %d" , pm[row][col]);
		while(row >= 0 && col >= 0)//pm[row][col] != 0)
		{
			//printf("\nNOW %d %d %d\n" , row , col , pm[row][col]);
			if(row == 0 && col == 0)
				break;
			if(pm[row][col] == 3)
			{
				r1[j] = key[row-1];
				r2[j] = '-';
				row -= 1;
				score += GP;
			}
			else if(pm[row][col] == 1)
			{
				r1[j] = '-';
				r2[j] = s[n*index + col-1];
				col -= 1;
				score += GP;
			}
			else if(pm[row][col] == 2)
			{
				r1[j] = key[row-1];
				r2[j] = s[n*index + col-1];
				if(key[row-1] == s[n*index + col-1])
					score += MR;
				else
					score += GP;
				row -= 1;
				col -= 1;
			}
			else
				score += GP;
			j += 1;
		}
		
		/*for(i = 0; i < j/2; i++)
		{
			r1[i] = (r1[i] + r1[j-i-1]) - (r1[j-i-1] = r1[i]);
			r2[i] = (r2[i] + r2[j-i-1]) - (r2[j-i-1] = r2[i]);
		} */
		
		r1[j] = '\0';
		r2[j] = '\0';
		printf("\nAlignment #%d :\n-------------------\nKey:\n%s\nQuery:\n%s\n" , index+1 , r1 , r2);
		
		scores[index] = score;
	}
}

int main(int argc, char** argv)
{
	int size = sizeof(int);
	int THREADS = 1024;
	
	freopen(argv[1] , "r", stdin);
	freopen(argv[2] , "w", stdout);
	
	int *host_scores , *scores;
	int i , num , n;
	
	//printf("Enter size:");
	scanf("%d" , &n);
	//printf("Enter number of queries:");
	scanf("%d" , &num);
	
	int m = num < THREADS ? num : THREADS;
	
	char *host_key = (char *)malloc(n);
	char *tmp = (char *)malloc(n);
	char *host_q = (char *)malloc(num * n + 2);
	char *key , *q;
	
	//printf("Enter key:");
	scanf("%s" , host_key);
	//printf("Enter the queries:");
	for(i = 0; i <num; i++)
	{
		if(i == 0)
			scanf("%s" , host_q);
		else
		{
			scanf("%s" , tmp);
			strcat(host_q , tmp);
		}
	}
	
	host_scores = (int *)malloc(size * num);
	
	hipMalloc((void **)&scores , num * size);
	hipMalloc((void **)&key , n);
	hipMalloc((void **)&q , n * num + 2);
	hipMemcpy(key , host_key , n , hipMemcpyHostToDevice);
	hipMemcpy(q , host_q , n * num + 2 , hipMemcpyHostToDevice);
	
	align <<<(n + m - 1) / m , m>>> (key , q , scores , n , num);

	hipMemcpy(host_scores , scores , size * num , hipMemcpyDeviceToHost);

	printf("\n\nAlignment Scores:\n----------------------------\n");
	for(i = 0; i < num; i++)
		printf("Query #%d : %d\n" , i+1 , host_scores[i]);
	hipFree(key);
	hipFree(q);
	hipFree(scores);
	return 0;
}