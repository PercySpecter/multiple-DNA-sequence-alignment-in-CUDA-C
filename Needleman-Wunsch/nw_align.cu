
/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 128

typedef long long int ll;

__global__ void align(char *key , char *s , ll *scores , ll n , ll num)
{
	ll GAP = -1 , MATCH = 1 , MISMATCH = -1;
	ll index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < num)
	{
		ll i , j , k , dia , top , left;
		ll dp[N + 1][N + 1];
		
		char r1[2*N+2] , r2[2*N+2];
		char traceback[N+1][N+1];
		for (i = 0; i <= n; i++)
		{
			dp[0][i] = GAP * i;
			dp[i][0] = GAP * i;
			traceback[0][i] = 'l';
			traceback[i][0] = 'u';
		}
		
		for (i = 1; i <= n; i++)
		{
			for (j = 1; j <= n; j++)
			{
				if(key[i-1] == s[n*index + j-1])
					dia = dp[i-1][j-1] + MATCH;
				else	
					dia = dp[i-1][j-1] + MISMATCH;
				top = dp[i-1][j] + GAP;
				left = dp[i][j-1] + GAP;
				dp[i][j] = dia > top ? (dia > left ? dia : left) : (top > left ? top : left);
				traceback[i][j] = dp[i][j] == dia ? 'd' : (dp[i][j] == top ? 'u' : 'l');
			}
		}
		/*
		for (i = 0; i <= n; i++)
		{
			for (j = 0; j <= n; j++)
			{
				printf("%lld " , dp[i][j]);
			}
			printf("\n");
		}
		for (i = 1; i <= n; i++)
		{
			for (j = 1; j <= n; j++)
			{
				printf("%c " , traceback[i][j]);
			}
			printf("\n");
		}
		*/
		i = n , j = n , k = 0;
		while(!(i == 0 && j == 0))
		{
			if(traceback[i][j] == 'd')
			{
				r1[k] = key[i-1];
				r2[k] = s[n*index + j-1];
				i--; 
				j--;
			}
			else if(traceback[i][j] == 'u')
			{
				r1[k] = key[i-1];
				r2[k] = '-';
				i--;
			}
			else
			{
				r1[k] = '-';
				r2[k] = s[n*index + j-1];
				j--;
			}
			k++;
		}
		for(i = 0; i < k/2; i++)
		{
			r1[i] = (r1[i] + r1[k-i-1]) - (r1[k-i-1] = r1[i]);
			r2[i] = (r2[i] + r2[k-i-1]) - (r2[k-i-1] = r2[i]);
		}
		r1[k] = '\0';
		r2[k] = '\0';
		printf("\nAlignment #%lld :\n%s\n%s\n" , index , r1 , r2);
		ll score = 0;
		for(i = 0; i < k; i++)
		{
			if(r1[k] == '-' || r2[k] == '-')
				score += GAP;
			else if(r1[i] == r2[i])
				score += MATCH;
			else
				score += MISMATCH;
		}
		scores[index] = score;
	}
}

int main(void)
{
	int size = sizeof(ll);
	ll THREADS = 1024;
	
	ll *host_scores , *scores;
	ll i , num , n;
	
	printf("Enter size:");
	scanf("%lld" , &n);
	printf("Enter number of queries:");
	scanf("%lld" , &num);
	
	char *host_key = (char *)malloc(n);
	char *tmp = (char *)malloc(n);
	char *host_q = (char *)malloc(num * n + 2);
	char *key , *q;
	
	printf("Enter key:");
	scanf("%s" , host_key);
	printf("Enter the queries:");
	for(i = 0; i <num; i++)
	{
		if(i == 0)
			scanf("%s" , host_q);
		else
		{
			scanf("%s" , tmp);
			strcat(host_q , tmp);
		}
	}
	
	host_scores = (ll *)malloc(size * num);
	
	hipMalloc((void **)&scores , num * size);
	hipMalloc((void **)&key , n);
	hipMalloc((void **)&q , n * num + 2);
	hipMemcpy(key , host_key , n , hipMemcpyHostToDevice);
	hipMemcpy(q , host_q , n * num + 2 , hipMemcpyHostToDevice);
	
	align <<<(n + THREADS - 1) / THREADS , THREADS>>> (key , q , scores , n , num);

	hipMemcpy(host_scores , scores , size * num , hipMemcpyDeviceToHost);

	printf("\n\nAlignment Scores:\n");
	for(i = 0; i < num; i++)
		printf("Query #%lld : %lld\n" , i , host_scores[i]);
	hipFree(key);
	hipFree(q);
	hipFree(scores);
	return 0;
}