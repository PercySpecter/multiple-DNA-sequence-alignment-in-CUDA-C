
/*#include "cuda_runtime.h"
#include "device_launch_parameters.h"*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 300

//typedef long long int ll;

__global__ void align(char *key , char *s , int *scores , int n , int num)
{
	int GAP = -1 , MATCH = 1 , MISMATCH = -1;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < num)
	{
		int i , j , k , dia , top , left;
		int dp[N + 1][N + 1];
		
		char r1[2*N+2] , r2[2*N+2];
		char traceback[N+1][N+1];
		for (i = 0; i <= n; i++)
		{
			dp[0][i] = GAP * i;
			dp[i][0] = GAP * i;
			traceback[0][i] = 'l';
			traceback[i][0] = 'u';
		}
		
		for (i = 1; i <= n; i++)
		{
			for (j = 1; j <= n; j++)
			{
				if(key[i-1] == s[n*index + j-1])
					dia = dp[i-1][j-1] + MATCH;
				else	
					dia = dp[i-1][j-1] + MISMATCH;
				top = dp[i-1][j] + GAP;
				left = dp[i][j-1] + GAP;
				dp[i][j] = dia > top ? (dia > left ? dia : left) : (top > left ? top : left);
				traceback[i][j] = dp[i][j] == dia ? 'd' : (dp[i][j] == top ? 'u' : 'l');
			}
		}
		/*
		for (i = 0; i <= n; i++)
		{
			for (j = 0; j <= n; j++)
			{
				printf("%d " , dp[i][j]);
			}
			printf("\n");
		}
		for (i = 1; i <= n; i++)
		{
			for (j = 1; j <= n; j++)
			{
				printf("%c " , traceback[i][j]);
			}
			printf("\n");
		}
		*/
		i = n , j = n , k = 0;
		while(!(i == 0 && j == 0))
		{
			if(traceback[i][j] == 'd')
			{
				r1[k] = key[i-1];
				r2[k] = s[n*index + j-1];
				i--; 
				j--;
			}
			else if(traceback[i][j] == 'u')
			{
				r1[k] = key[i-1];
				r2[k] = '-';
				i--;
			}
			else
			{
				r1[k] = '-';
				r2[k] = s[n*index + j-1];
				j--;
			}
			k++;
		}
		for(i = 0; i < k/2; i++)
		{
			r1[i] = (r1[i] + r1[k-i-1]) - (r1[k-i-1] = r1[i]);
			r2[i] = (r2[i] + r2[k-i-1]) - (r2[k-i-1] = r2[i]);
		}
		r1[k] = '\0';
		r2[k] = '\0';
		printf("\nAlignment #%d :\n%s\n%s\n" , index+1 , r1 , r2);
		int score = 0;
		for(i = 0; i < k; i++)
		{
			if(r1[k] == '-' || r2[k] == '-')
				score += GAP;
			else if(r1[i] == r2[i])
				score += MATCH;
			else
				score += MISMATCH;
		}
		scores[index] = score;
	}
}

int main(void)
{
	int size = sizeof(int);
	int THREADS = 1024;
	
	int *host_scores , *scores;
	int i , num , n;
	
	printf("Enter size:");
	scanf("%d" , &n);
	printf("Enter number of queries:");
	scanf("%d" , &num);
	
	int m = n < THREADS ? n : THREADS;
	
	char *host_key = (char *)malloc(n);
	char *tmp = (char *)malloc(n);
	char *host_q = (char *)malloc(num * n + 2);
	char *key , *q;
	
	printf("Enter key:");
	scanf("%s" , host_key);
	printf("Enter the queries:");
	for(i = 0; i <num; i++)
	{
		if(i == 0)
			scanf("%s" , host_q);
		else
		{
			scanf("%s" , tmp);
			strcat(host_q , tmp);
		}
	}
	
	host_scores = (int *)malloc(size * num);
	
	hipMalloc((void **)&scores , num * size);
	hipMalloc((void **)&key , n);
	hipMalloc((void **)&q , n * num + 2);
	hipMemcpy(key , host_key , n , hipMemcpyHostToDevice);
	hipMemcpy(q , host_q , n * num + 2 , hipMemcpyHostToDevice);
	
	align <<<(n + m - 1) / m , m>>> (key , q , scores , n , num);

	hipMemcpy(host_scores , scores , size * num , hipMemcpyDeviceToHost);

	printf("\n\nAlignment Scores:\n");
	for(i = 0; i < num; i++)
		printf("Query #%d : %d\n" , i+1 , host_scores[i]);
	hipFree(key);
	hipFree(q);
	hipFree(scores);
	return 0;
}